#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <sys/time.h>
#include "layer.h"
#define DEBUG_MODE 0
#define CPU_DEBUG_MODE 0
using namespace std;

enum LAYER
{
    CONV2D_1,
    CONV2D_2,
    MAXP2D_1,
    CONV2D_3,
    CONV2D_4,
    MAXP2D_2,
    CONV2D_5,
    CONV2D_6,
    MAXP2D_3,
    CONV2D_7,
    CONV2D_8,
    MAXP2D_4,
    CONV2D_9,
    CONV2D_10,
    UPSM2D_1,
    CONV2D_11,
    CONCAT_1,
    CONV2D_12,
    CONV2D_13,
    UPSM2D_2,
    CONV2D_14,
    CONCAT_2,
    CONV2D_15,
    CONV2D_16,
    UPSM2D_3,
    CONV2D_17,
    CONCAT_3,
    CONV2D_18,
    CONV2D_19,
    UPSM2D_4,
    CONV2D_20,
    CONCAT_4,
    CONV2D_21,
    CONV2D_22,
    CONV2D_23,
    CONV2D_24,


};

float time_sum=0;
//global variables
//*******conv2d_1**********
float* d_input{0};
float* d_output{0};
float* d_kernel{0};
float* d_bias{0};
//*******conv2d_2**********
float* d_output_2{0};
float* d_kernel_2{0};
float* d_bias_2{0};
//******max_pooling2d_1****
float* d_output_maxp_1{0};
//*******conv2d_3**********
float* d_output_3{0};
float* d_kernel_3{0};
float* d_bias_3{0};
//*******conv2d_4**********
float* d_output_4{0};
float* d_kernel_4{0};
float* d_bias_4{0};
//******max_pooling2d_2****
float* d_output_maxp_2{0};
//*******conv2d_5**********
float* d_output_5{0};
float* d_kernel_5{0};
float* d_bias_5{0};
//*******conv2d_6**********
float* d_output_6{0};
float* d_kernel_6{0};
float* d_bias_6{0};
//******max_pooling2d_3****
float* d_output_maxp_3{0};
//*******conv2d_7**********
float* d_output_7{0};
float* d_kernel_7{0};
float* d_bias_7{0};
//*******conv2d_8**********
float* d_output_8{0};
float* d_kernel_8{0};
float* d_bias_8{0};
//******max_pooling2d_4****
float* d_output_maxp_4{0};
//*******conv2d_9**********
float* d_output_9{0};
float* d_kernel_9{0};
float* d_bias_9{0};
//*******conv2d_10**********
float* d_output_10{0};
float* d_kernel_10{0};
float* d_bias_10{0};
//******max_pooling2d_4****
float* d_output_upsm_1{0};
//*******conv2d_11**********
float* d_output_11{0};
float* d_kernel_11{0};
float* d_bias_11{0};
//******concat_1****
float* d_output_concat_1{0};
//*******conv2d_12**********
float* d_output_12{0};
float* d_kernel_12{0};
float* d_bias_12{0};
//*******conv2d_13**********
float* d_output_13{0};
float* d_kernel_13{0};
float* d_bias_13{0};
//******upsample_2d_2****
float* d_output_upsm_2{0};
//*******conv2d_14**********
float* d_output_14{0};
float* d_kernel_14{0};
float* d_bias_14{0};
//******concat_2****
float* d_output_concat_2{0};
//*******conv2d_15**********
float* d_output_15{0};
float* d_kernel_15{0};
float* d_bias_15{0};
//*******conv2d_16**********
float* d_output_16{0};
float* d_kernel_16{0};
float* d_bias_16{0};
//******upsample_2d_3****
float* d_output_upsm_3{0};
//*******conv2d_17**********
float* d_output_17{0};
float* d_kernel_17{0};
float* d_bias_17{0};
//******concat_3****
float* d_output_concat_3{0};
//*******conv2d_18**********
float* d_output_18{0};
float* d_kernel_18{0};
float* d_bias_18{0};
//*******conv2d_19**********
float* d_output_19{0};
float* d_kernel_19{0};
float* d_bias_19{0};
//******upsample_2d_4****
float* d_output_upsm_4{0};
//*******conv2d_20**********
float* d_output_20{0};
float* d_kernel_20{0};
float* d_bias_20{0};
//******concat_4****
float* d_output_concat_4{0};
//*******conv2d_21**********
float* d_output_21{0};
float* d_kernel_21{0};
float* d_bias_21{0};
//*******conv2d_22**********
float* d_output_22{0};
float* d_kernel_22{0};
float* d_bias_22{0};
//*******conv2d_23**********
float* d_output_23{0};
float* d_kernel_23{0};
float* d_bias_23{0};
//*******conv2d_24**********
float* d_output_24{0};
float* d_kernel_24{0};
float* d_bias_24{0};

#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
#define CHECK(call){gpu_error_checker((call),__FILE__,__LINE__);}

inline void gpu_error_checker(hipError_t error,const char*file, int line)
{
    if (error != hipSuccess)
    {
        printf("Error:file %s: line %d, ", __FILE__, __LINE__);
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error));
    }
}

double GetTime() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

__device__ float relu(float input)
{
    if(input<0)
    {
        input=0;
    }
    return input;
}
__global__ void CONV2DGPU1(float *input_image, float *output_image, float *Kernel,float *bias,
                           int nx, int ny,int layer) {
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx_base = iy*nx + ix;
    if (ix < nx && iy < ny)
    {
        float sum=0;
        int which_layer=layer*9;
        for (char r = -1; r < 2; ++r)
        {
            for (char c = -1; c < 2; ++c)
            {
                unsigned int idx = (iy+r)*nx + (ix+c);
                if(!(iy+r==-1 | ix+c==-1 | ix+c==nx | iy+r==ny ))
                {
                    sum+=input_image[idx]*Kernel[3*(r+1)+(c+1)+which_layer];
                    //#if DEBUG_MODE==1
                    //                    printf("row:%d col:%d Pixel:%f Kernel:%f SUM:%f\n",iy+r,
                    //                           ix+c,input_image[idx],Kernel[3*(r+1)+(c+1)+layer*9],sum);
                    //#endif
                }
            }
        }
        output_image[idx_base+nx*ny*layer]=relu(sum+bias[layer]);
    }
}
__global__ void CONV2D_2_GPU1(float *input_image, float *output_image, float *Kernel,float *bias,
                              int nx, int ny,int layer,int depth)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx_base = iy*nx + ix;
    if (ix < nx && iy < ny /*&& ix==0 && iy==0*/)
    {
        float sum=0;
        for (int dth = 0; dth < depth; ++dth)    //dth---> depth
        {
            int depth_index=dth*nx*ny;
            for (char r = -1; r < 2; ++r)
            {
                for (char c = -1; c < 2; ++c)
                {
                    unsigned int idx = (iy+r)*nx + (ix+c) + depth_index;
                    if(!(iy+r==-1 | ix+c==-1 | ix+c==nx | iy+r==ny ))
                    {
                        sum+=input_image[idx]*Kernel[3*(r+1)+(c+1)+dth*9+layer*9*depth];
#if DEBUG_MODE==1
                        printf("depth:%d row:%d col:%d Pixel:%f Kernel:%f SUM:%f\n",dth,iy+r,
                               ix+c,input_image[idx],Kernel[3*(r+1)+(c+1)+dth*9+layer*9*16],sum);
#endif
                    }
                }
            }
#if DEBUG_MODE==1
            printf("\033[1;31m-------------------------------------------------\033[0m\n");
#endif
        }
        output_image[idx_base+nx*ny*layer]=relu(sum+bias[layer]);
#if DEBUG_MODE==1
        printf("\033[1;32msum:%f\033[0m\n",output_image[idx_base+nx*ny*layer]);
#endif
    }
}


__global__ void MAXP2D_GPU(float *input_image, float *output_image,int nx, int ny,int layer)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx_base = iy*nx + ix + layer*nx*ny;
    if (ix < nx && iy < ny)
    {
        unsigned int x_index=2*ix;
        unsigned int offset=layer*2*nx*2*ny;

        unsigned int idx_1= (x_index)+(   (2*iy)*(2*nx)  )+offset;  //top-left
        unsigned int idx_2= (x_index+1)+(   (2*iy)*(2*nx)  )+offset;  //top-right
        unsigned int idx_3= (x_index)+(   (2*iy+1)*(2*nx)  )+offset;  //bottom-left
        unsigned int idx_4= (x_index+1)+(   (2*iy+1)*(2*nx)  )+offset;  //bottom-right
        float max;
        if(input_image[idx_1]>input_image[idx_2])
        {
            max=input_image[idx_1];
        }
        else
        {
            max=input_image[idx_2];
        }

        if(max<input_image[idx_3])
        {
            max=input_image[idx_3];
        }

        if(max<input_image[idx_4])
        {
            max=input_image[idx_4];
        }
        output_image[idx_base]=max;
        //        printf("idx:%d = %f\n",idx_base,output_image[idx_base]);
        //        printf("ix:%d iy:%d index number:%d tl:%f tr:%f bl:%f br:%f output:%f \n",ix,iy,idx_base,
        //               input_image[idx_1],input_image[idx_2],input_image[idx_3],input_image[idx_4],output_image[idx_base]);
    }
}

__global__ void UPSM_2D_GPU(float *input_image, float *output_image,int nx, int ny,int layer)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx_base = iy*nx + ix + layer*nx*ny;
    if (ix < nx && iy < ny)
    {
        unsigned int x_index=2*ix;
        unsigned int offset=layer*2*nx*2*ny;

        unsigned int idx_1= (x_index)+(   (2*iy)*(2*nx)  )+offset;  //top-left
        unsigned int idx_2= (x_index+1)+(   (2*iy)*(2*nx)  )+offset;  //top-right
        unsigned int idx_3= (x_index)+(   (2*iy+1)*(2*nx)  )+offset;  //bottom-left
        unsigned int idx_4= (x_index+1)+(   (2*iy+1)*(2*nx)  )+offset;  //bottom-right
        output_image[idx_1]=input_image[idx_base];
        output_image[idx_2]=input_image[idx_base];
        output_image[idx_3]=input_image[idx_base];
        output_image[idx_4]=input_image[idx_base];
    }
}

__global__ void CONCAT_GPU(float *input_image1,  //first volume
                            float *input_image2,  //second volume
                            float *output_image,  //concat first volume and second volume
                            int nx,
                            int ny,
                            int layer,            //which layer is under process
                            int NLayer )          //number of total layers
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx_base1 = iy*nx + ix + layer*nx*ny;
    unsigned int idx_base2 = iy*nx + ix + (NLayer/2+layer)*nx*ny;
    if (ix < nx && iy < ny)
    {
        output_image[idx_base1]=input_image1[idx_base1];
        output_image[idx_base2]=input_image2[idx_base1];
//        printf("ix:%d, iy:%d, index:%d, pixel1:%f,pixel2:%f\n",ix,iy,idx_base1,input_image1[idx_base1],input_image2[idx_base1]);
    }
}

extern "C" void conv2d_1(float* img_ptr,float** output,int w,int h,layer l)
{
    time_sum=0;
    hipMemcpy(d_input, img_ptr, l.input_size, hipMemcpyHostToDevice);
    hipMemset(d_output, 0, l.output_size);
    CHECK(hipMemcpy(d_kernel,l.weight,l.kernel_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_bias,l.bias,l.bias_size,hipMemcpyHostToDevice));

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)
    {
        CONV2DGPU1<<<grid,block>>>(d_input,d_output,d_kernel,d_bias,w,h,i);
    }
    printf("time elapsed \033[1;33mconv2d_1:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output, l.output_size, hipMemcpyDeviceToHost);

    *output=h_output;

}

extern "C" void conv2d_2(float** output, int w, int h, layer l)
{
#if CPU_DEBUG_MODE==1
    for (int i = 0; i < 10; ++i)
    {
        printf("line:%d conv2d_2 output[%d]:%f\n",__LINE__,i,img_ptr[i*w*h]);
    }
#endif
    hipMemset(d_output_2, 0, l.output_size);
    CHECK(hipMemcpy(d_kernel_2,l.weight,l.kernel_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_bias_2,l.bias,l.bias_size,hipMemcpyHostToDevice));

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        CONV2D_2_GPU1<<<grid,block>>>(d_output,d_output_2,d_kernel_2,d_bias_2,w,h,i,l.depth);
    }
    printf("time elapsed \033[1;33mconv2d_2:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_2,l.output_size, hipMemcpyDeviceToHost);

    *output=h_output;
}

extern "C" void maxp2d_1(float** output, int w, int h, layer l)
{
    hipMemset(d_output_maxp_1, 0, l.output_size);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        MAXP2D_GPU<<<grid,block>>>(d_output_2,d_output_maxp_1,w,h,i);
    }
    printf("time elapsed \033[1;33mmaxp2d_1:%f msec \n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_maxp_1,l.output_size, hipMemcpyDeviceToHost);
    *output=h_output;
}
extern "C" void conv2d_3(float** output, int w, int h, layer l)
{
#if CPU_DEBUG_MODE==1
    for (int i = 0; i < 10; ++i)
    {
        printf("line:%d conv2d_2 output[%d]:%f\n",__LINE__,i,img_ptr[i*w*h]);
    }
#endif
    hipMemset(d_output_3, 0, l.output_size);
    CHECK(hipMemcpy(d_kernel_3,l.weight,l.kernel_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_bias_3,l.bias,l.bias_size,hipMemcpyHostToDevice));

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        CONV2D_2_GPU1<<<grid,block>>>(d_output_maxp_1,d_output_3,d_kernel_3,d_bias_3,
                                       w,h,i,l.depth);
    }
    printf("time elapsed \033[1;33mconv2d_3:%f msec \n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_3,l.output_size, hipMemcpyDeviceToHost);

    *output=h_output;
}

extern "C" void conv2d_4(float** output, int w, int h, layer l)
{
#if CPU_DEBUG_MODE==1
    for (int i = 0; i < 10; ++i)
    {
        printf("line:%d conv2d_2 output[%d]:%f\n",__LINE__,i,img_ptr[i*w*h]);
    }
#endif
    hipMemset(d_output_4, 0, l.output_size);
    CHECK(hipMemcpy(d_kernel_4,l.weight,l.kernel_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_bias_4,l.bias,l.bias_size,hipMemcpyHostToDevice));

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        CONV2D_2_GPU1<<<grid,block>>>(d_output_3,d_output_4,d_kernel_4,d_bias_4,
                                       w,h,i,l.depth);
    }
    printf("time elapsed \033[1;33mconv2d_4:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_4,l.output_size, hipMemcpyDeviceToHost);

    *output=h_output;
}

extern "C" void maxp2d_2(float** output, int w, int h, layer l)
{
    hipMemset(d_output_maxp_2, 0, l.output_size);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        MAXP2D_GPU<<<grid,block>>>(d_output_4,d_output_maxp_2,w,h,i);
    }
    printf("time elapsed \033[1;33mmaxp2d_2:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_maxp_2,l.output_size, hipMemcpyDeviceToHost);
    *output=h_output;
}
extern "C" void conv2d_5(float** output, int w, int h, layer l)
{
    hipMemset(d_output_5, 0, l.output_size);
    CHECK(hipMemcpy(d_kernel_5,l.weight,l.kernel_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_bias_5,l.bias,l.bias_size,hipMemcpyHostToDevice));

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        CONV2D_2_GPU1<<<grid,block>>>(d_output_maxp_2,d_output_5,d_kernel_5,d_bias_5,
                                       w,h,i,l.depth);
    }
    printf("time elapsed \033[1;33mconv2d_5:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_5,l.output_size, hipMemcpyDeviceToHost);

    *output=h_output;
}

extern "C" void conv2d_6(float** output, int w, int h, layer l)
{
    hipMemset(d_output_6, 0, l.output_size);
    CHECK(hipMemcpy(d_kernel_6,l.weight,l.kernel_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_bias_6,l.bias,l.bias_size,hipMemcpyHostToDevice));

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        CONV2D_2_GPU1<<<grid,block>>>(d_output_5,d_output_6,d_kernel_6,d_bias_6,
                                       w,h,i,l.depth);
    }
    printf("time elapsed \033[1;33mconv2d_6:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_6,l.output_size, hipMemcpyDeviceToHost);

    *output=h_output;
}

extern "C" void maxp2d_3(float** output, int w, int h, layer l)
{
    hipMemset(d_output_maxp_3, 0, l.output_size);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        MAXP2D_GPU<<<grid,block>>>(d_output_6,d_output_maxp_3,w,h,i);
    }
    printf("time elapsed \033[1;33mmaxp2d_2:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_maxp_3,l.output_size, hipMemcpyDeviceToHost);
    *output=h_output;
}

extern "C" void conv2d_7(float** output, int w, int h, layer l)
{
    hipMemset(d_output_7, 0, l.output_size);
    CHECK(hipMemcpy(d_kernel_7,l.weight,l.kernel_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_bias_7,l.bias,l.bias_size,hipMemcpyHostToDevice));

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        CONV2D_2_GPU1<<<grid,block>>>(d_output_maxp_3,d_output_7,d_kernel_7,d_bias_7,
                                       w,h,i,l.depth);
    }
    printf("time elapsed \033[1;33mconv2d_7:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_7,l.output_size, hipMemcpyDeviceToHost);

    *output=h_output;
}

extern "C" void conv2d_8(float** output, int w, int h, layer l)
{
    hipMemset(d_output_8, 0, l.output_size);
    CHECK(hipMemcpy(d_kernel_8,l.weight,l.kernel_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_bias_8,l.bias,l.bias_size,hipMemcpyHostToDevice));

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        CONV2D_2_GPU1<<<grid,block>>>(d_output_7,d_output_8,d_kernel_8,d_bias_8,
                                       w,h,i,l.depth);
    }
    printf("time elapsed \033[1;33mconv2d_7:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_8,l.output_size, hipMemcpyDeviceToHost);

    *output=h_output;
}

extern "C" void maxp2d_4(float** output, int w, int h, layer l)
{
    hipMemset(d_output_maxp_4, 0, l.output_size);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        MAXP2D_GPU<<<grid,block>>>(d_output_8,d_output_maxp_4,w,h,i);
    }
    printf("time elapsed \033[1;33mmaxp2d_2:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_maxp_4,l.output_size, hipMemcpyDeviceToHost);
    *output=h_output;
}

extern "C" void conv2d_9(float** output, int w, int h, layer l)
{
    hipMemset(d_output_9, 0, l.output_size);
    CHECK(hipMemcpy(d_kernel_9,l.weight,l.kernel_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_bias_9,l.bias,l.bias_size,hipMemcpyHostToDevice));

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        CONV2D_2_GPU1<<<grid,block>>>(d_output_maxp_4,d_output_9,d_kernel_9,d_bias_9,
                                       w,h,i,l.depth);
    }
    printf("time elapsed \033[1;33mconv2d_9:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_9,l.output_size, hipMemcpyDeviceToHost);

    *output=h_output;
}

extern "C" void conv2d_10(float** output, int w, int h, layer l)
{
    hipMemset(d_output_10, 0, l.output_size);
    CHECK(hipMemcpy(d_kernel_10,l.weight,l.kernel_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_bias_10,l.bias,l.bias_size,hipMemcpyHostToDevice));

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        CONV2D_2_GPU1<<<grid,block>>>(d_output_9,d_output_10,d_kernel_10,d_bias_10,
                                       w,h,i,l.depth);
    }
    printf("time elapsed \033[1;33mconv2d_10:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_10,l.output_size, hipMemcpyDeviceToHost);

    *output=h_output;
}

extern "C" void upsample_2d_1(float** output, int w, int h, layer l)
{
    hipMemset(d_output_upsm_1, 0, l.output_size);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w/2 + block.x - 1) / block.x, (h/2 + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        UPSM_2D_GPU<<<grid,block>>>(d_output_10,d_output_upsm_1,w/2,h/2,i);
    }
    printf("time elapsed \033[1;33mupsample_1:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_upsm_1,l.output_size, hipMemcpyDeviceToHost);
    *output=h_output;
}

extern "C" void conv2d_11(float** output, int w, int h, layer l)
{
    hipMemset(d_output_11, 0, l.output_size);
    CHECK(hipMemcpy(d_kernel_11,l.weight,l.kernel_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_bias_11,l.bias,l.bias_size,hipMemcpyHostToDevice));

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        CONV2D_2_GPU1<<<grid,block>>>(d_output_upsm_1,d_output_11,d_kernel_11,d_bias_11,
                                       w,h,i,l.depth);
    }
    printf("time elapsed \033[1;33mconv2d_11:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_11,l.output_size, hipMemcpyDeviceToHost);

    *output=h_output;
}

extern "C" void concat_1(float** output, int w, int h, layer l)
{
    hipMemset(d_output_concat_1, 0, l.output_size);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters/2; ++i)   //which volume is selected for conv
    {
        CONCAT_GPU<<<grid,block>>>(d_output_8,d_output_11,d_output_concat_1,w,h,i,l.nfilters);
    }
    printf("time elapsed \033[1;33mupsample_1:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_concat_1,l.output_size, hipMemcpyDeviceToHost);
    *output=h_output;
}

extern "C" void conv2d_12(float** output, int w, int h, layer l)
{
    hipMemset(d_output_12, 0, l.output_size);
    CHECK(hipMemcpy(d_kernel_12,l.weight,l.kernel_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_bias_12,l.bias,l.bias_size,hipMemcpyHostToDevice));

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        CONV2D_2_GPU1<<<grid,block>>>(d_output_concat_1,d_output_12,d_kernel_12,d_bias_12,
                                       w,h,i,l.depth);
    }
    printf("time elapsed \033[1;33mconv2d_11:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_12,l.output_size, hipMemcpyDeviceToHost);

    *output=h_output;
}

extern "C" void conv2d_13(float** output, int w, int h, layer l)
{
    hipMemset(d_output_13, 0, l.output_size);
    CHECK(hipMemcpy(d_kernel_13,l.weight,l.kernel_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_bias_13,l.bias,l.bias_size,hipMemcpyHostToDevice));

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        CONV2D_2_GPU1<<<grid,block>>>(d_output_12,d_output_13,d_kernel_13,d_bias_13,
                                       w,h,i,l.depth);
    }
    printf("time elapsed \033[1;33mconv2d_11:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_13,l.output_size, hipMemcpyDeviceToHost);

    *output=h_output;
}

extern "C" void upsample_2d_2(float** output, int w, int h, layer l)
{
    hipMemset(d_output_upsm_2, 0, l.output_size);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w/2 + block.x - 1) / block.x, (h/2 + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        UPSM_2D_GPU<<<grid,block>>>(d_output_13,d_output_upsm_2,w/2,h/2,i);
    }
    printf("time elapsed \033[1;33mupsample_2:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_upsm_2,l.output_size, hipMemcpyDeviceToHost);
    *output=h_output;
}

extern "C" void conv2d_14(float** output, int w, int h, layer l)
{
    hipMemset(d_output_14, 0, l.output_size);
    CHECK(hipMemcpy(d_kernel_14,l.weight,l.kernel_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_bias_14,l.bias,l.bias_size,hipMemcpyHostToDevice));

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        CONV2D_2_GPU1<<<grid,block>>>(d_output_upsm_2,d_output_14,d_kernel_14,d_bias_14,
                                       w,h,i,l.depth);
    }
    printf("time elapsed \033[1;33mconv2d_14:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_14,l.output_size, hipMemcpyDeviceToHost);

    *output=h_output;
}

extern "C" void concat_2(float** output, int w, int h, layer l)
{
    hipMemset(d_output_concat_2, 0, l.output_size);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters/2; ++i)   //which volume is selected for conv
    {
        CONCAT_GPU<<<grid,block>>>(d_output_6,d_output_14,d_output_concat_2,w,h,i,l.nfilters);
    }
    printf("time elapsed \033[1;33mupsample_1:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_concat_2,l.output_size, hipMemcpyDeviceToHost);
    *output=h_output;
}

extern "C" void conv2d_15(float** output, int w, int h, layer l)
{
    hipMemset(d_output_15, 0, l.output_size);
    CHECK(hipMemcpy(d_kernel_15,l.weight,l.kernel_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_bias_15,l.bias,l.bias_size,hipMemcpyHostToDevice));

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        CONV2D_2_GPU1<<<grid,block>>>(d_output_concat_2,d_output_15,d_kernel_15,d_bias_15,
                                       w,h,i,l.depth);
    }
    printf("time elapsed \033[1;33mconv2d_14:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_15,l.output_size, hipMemcpyDeviceToHost);

    *output=h_output;
}

extern "C" void conv2d_16(float** output, int w, int h, layer l)
{
    hipMemset(d_output_16, 0, l.output_size);
    CHECK(hipMemcpy(d_kernel_16,l.weight,l.kernel_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_bias_16,l.bias,l.bias_size,hipMemcpyHostToDevice));

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        CONV2D_2_GPU1<<<grid,block>>>(d_output_15,d_output_16,d_kernel_16,d_bias_16,
                                       w,h,i,l.depth);
    }
    printf("time elapsed \033[1;33mconv2d_14:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_16,l.output_size, hipMemcpyDeviceToHost);

    *output=h_output;
}
extern "C" void upsample_2d_3(float** output, int w, int h, layer l)
{
    hipMemset(d_output_upsm_3, 0, l.output_size);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w/2 + block.x - 1) / block.x, (h/2 + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        UPSM_2D_GPU<<<grid,block>>>(d_output_16,d_output_upsm_3,w/2,h/2,i);
    }
    printf("time elapsed \033[1;33mupsample_2:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_upsm_3,l.output_size, hipMemcpyDeviceToHost);
    *output=h_output;
}
extern "C" void conv2d_17(float** output, int w, int h, layer l)
{
    hipMemset(d_output_17, 0, l.output_size);
    CHECK(hipMemcpy(d_kernel_17,l.weight,l.kernel_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_bias_17,l.bias,l.bias_size,hipMemcpyHostToDevice));

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        CONV2D_2_GPU1<<<grid,block>>>(d_output_upsm_3,d_output_17,d_kernel_17,d_bias_17,
                                       w,h,i,l.depth);
    }
    printf("time elapsed \033[1;33mconv2d_14:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_17,l.output_size, hipMemcpyDeviceToHost);

    *output=h_output;
}

extern "C" void concat_3(float** output, int w, int h, layer l)
{
    hipMemset(d_output_concat_3, 0, l.output_size);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters/2; ++i)   //which volume is selected for conv
    {
        CONCAT_GPU<<<grid,block>>>(d_output_4,d_output_17,d_output_concat_3,w,h,i,l.nfilters);
    }
    printf("time elapsed \033[1;33mupsample_1:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_concat_3,l.output_size, hipMemcpyDeviceToHost);
    *output=h_output;
}
extern "C" void conv2d_18(float** output, int w, int h, layer l)
{
    hipMemset(d_output_18, 0, l.output_size);
    CHECK(hipMemcpy(d_kernel_18,l.weight,l.kernel_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_bias_18,l.bias,l.bias_size,hipMemcpyHostToDevice));

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        CONV2D_2_GPU1<<<grid,block>>>(d_output_concat_3,d_output_18,d_kernel_18,d_bias_18,
                                       w,h,i,l.depth);
    }
    printf("time elapsed \033[1;33mconv2d_14:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_18,l.output_size, hipMemcpyDeviceToHost);

    *output=h_output;
}
extern "C" void conv2d_19(float** output, int w, int h, layer l)
{
    hipMemset(d_output_19, 0, l.output_size);
    CHECK(hipMemcpy(d_kernel_19,l.weight,l.kernel_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_bias_19,l.bias,l.bias_size,hipMemcpyHostToDevice));

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        CONV2D_2_GPU1<<<grid,block>>>(d_output_18,d_output_19,d_kernel_19,d_bias_19,
                                       w,h,i,l.depth);
    }
    printf("time elapsed \033[1;33mconv2d_14:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_19,l.output_size, hipMemcpyDeviceToHost);

    *output=h_output;
}
extern "C" void upsample_2d_4(float** output, int w, int h, layer l)
{
    hipMemset(d_output_upsm_4, 0, l.output_size);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w/2 + block.x - 1) / block.x, (h/2 + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        UPSM_2D_GPU<<<grid,block>>>(d_output_19,d_output_upsm_4,w/2,h/2,i);
    }
    printf("time elapsed \033[1;33mupsample_2:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_upsm_4,l.output_size, hipMemcpyDeviceToHost);
    *output=h_output;
}
extern "C" void conv2d_20(float** output, int w, int h, layer l)
{
    hipMemset(d_output_20, 0, l.output_size);
    CHECK(hipMemcpy(d_kernel_20,l.weight,l.kernel_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_bias_20,l.bias,l.bias_size,hipMemcpyHostToDevice));

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        CONV2D_2_GPU1<<<grid,block>>>(d_output_upsm_4,d_output_20,d_kernel_20,d_bias_20,
                                       w,h,i,l.depth);
    }
    printf("time elapsed \033[1;33mconv2d_20:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_20,l.output_size, hipMemcpyDeviceToHost);

    *output=h_output;
}

extern "C" void concat_4(float** output, int w, int h, layer l)
{
    hipMemset(d_output_concat_4, 0, l.output_size);

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters/2; ++i)   //which volume is selected for conv
    {
        CONCAT_GPU<<<grid,block>>>(d_output_2,d_output_20,d_output_concat_4,w,h,i,l.nfilters);
    }
    printf("time elapsed \033[1;33mupsample_1:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_concat_4,l.output_size, hipMemcpyDeviceToHost);
    *output=h_output;
}
extern "C" void conv2d_21(float** output, int w, int h, layer l)
{
    hipMemset(d_output_21, 0, l.output_size);
    CHECK(hipMemcpy(d_kernel_21,l.weight,l.kernel_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_bias_21,l.bias,l.bias_size,hipMemcpyHostToDevice));

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        CONV2D_2_GPU1<<<grid,block>>>(d_output_concat_4,d_output_21,d_kernel_21,d_bias_21,
                                       w,h,i,l.depth);
    }
    printf("time elapsed \033[1;33mconv2d_20:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_21,l.output_size, hipMemcpyDeviceToHost);

    *output=h_output;
}
extern "C" void conv2d_22(float** output, int w, int h, layer l)
{
    hipMemset(d_output_22, 0, l.output_size);
    CHECK(hipMemcpy(d_kernel_22,l.weight,l.kernel_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_bias_22,l.bias,l.bias_size,hipMemcpyHostToDevice));

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        CONV2D_2_GPU1<<<grid,block>>>(d_output_21,d_output_22,d_kernel_22,d_bias_22,
                                       w,h,i,l.depth);
    }
    printf("time elapsed \033[1;33mconv2d_20:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_22,l.output_size, hipMemcpyDeviceToHost);

    *output=h_output;
}
extern "C" void conv2d_23(float** output, int w, int h, layer l)
{
    hipMemset(d_output_23, 0, l.output_size);
    CHECK(hipMemcpy(d_kernel_23,l.weight,l.kernel_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_bias_23,l.bias,l.bias_size,hipMemcpyHostToDevice));

    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
    double t1=GetTime();
    for (int i = 0; i < l.nfilters; ++i)   //which volume is selected for conv
    {
        CONV2D_2_GPU1<<<grid,block>>>(d_output_22,d_output_23,d_kernel_23,d_bias_23,
                                       w,h,i,l.depth);
    }
    printf("time elapsed \033[1;33mconv2d_20:%f msec\n\033[0m",1000*(GetTime()-t1));
    float* h_output =(float*)malloc(l.output_size);
    hipMemcpy(h_output, d_output_23,l.output_size, hipMemcpyDeviceToHost);

    *output=h_output;
}
extern "C" void LOAD_NEURAL_NETWORK(LAYER Layer, int w, int h, layer& l)
{
    switch (Layer) {
    case CONV2D_1:
    {
        l.input_size = w * h * sizeof(float);
        hipMalloc(&d_input, l.input_size);

        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output, l.output_size);

        l.kernel_size = l.width * l.height * l.nfilters * sizeof(float);
        hipMalloc((void**)&d_kernel,l.kernel_size);

        l.bias_size = l.nfilters * sizeof(float);
        hipMalloc((void**)&d_bias,l.bias_size);

        l.im_h=h;
        l.im_w=w;
        printf("\033[1;31mLOAD CONV2D_1: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case CONV2D_2:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_2, l.output_size);

        l.kernel_size = l.width * l.height * l.nfilters * l.depth * sizeof(float);
        hipMalloc( (void**)&d_kernel_2, l.kernel_size);

        l.bias_size = l.nfilters * sizeof(float);
        hipMalloc((void**)&d_bias_2, l.bias_size);

        l.im_h=h;
        l.im_w=w;
        printf("\033[1;31mLOAD CONV2D_2: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case MAXP2D_1:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_maxp_1, l.output_size);

        l.im_h=h;
        l.im_w=w;

        printf("\033[1;31mLOAD MAXP2D_1: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case CONV2D_3:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_3, l.output_size);

        l.kernel_size = l.width * l.height * l.nfilters * l.depth * sizeof(float);
        hipMalloc( (void**)&d_kernel_3, l.kernel_size);

        l.bias_size = l.nfilters * sizeof(float);
        hipMalloc((void**)&d_bias_3, l.bias_size);

        l.im_h=h;
        l.im_w=w;
        printf("\033[1;31mLOAD CONV2D_3: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case CONV2D_4:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_4, l.output_size);

        l.kernel_size = l.width * l.height * l.nfilters * l.depth * sizeof(float);
        hipMalloc( (void**)&d_kernel_4, l.kernel_size);

        l.bias_size = l.nfilters * sizeof(float);
        hipMalloc((void**)&d_bias_4, l.bias_size);

        l.im_h=h;
        l.im_w=w;
        printf("\033[1;31mLOAD CONV2D_4: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case MAXP2D_2:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_maxp_2, l.output_size);

        l.im_h=h;
        l.im_w=w;

        printf("\033[1;31mLOAD MAXP2D_2: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case CONV2D_5:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_5, l.output_size);

        l.kernel_size = l.width * l.height * l.nfilters * l.depth * sizeof(float);
        hipMalloc( (void**)&d_kernel_5, l.kernel_size);

        l.bias_size = l.nfilters * sizeof(float);
        hipMalloc((void**)&d_bias_5, l.bias_size);

        l.im_h=h;
        l.im_w=w;
        printf("\033[1;31mLOAD CONV2D_5: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case CONV2D_6:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_6, l.output_size);

        l.kernel_size = l.width * l.height * l.nfilters * l.depth * sizeof(float);
        hipMalloc( (void**)&d_kernel_6, l.kernel_size);

        l.bias_size = l.nfilters * sizeof(float);
        hipMalloc((void**)&d_bias_6, l.bias_size);

        l.im_h=h;
        l.im_w=w;
        printf("\033[1;31mLOAD CONV2D_6: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case MAXP2D_3:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_maxp_3, l.output_size);

        l.im_h=h;
        l.im_w=w;

        printf("\033[1;31mLOAD MAXP2D_3: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case CONV2D_7:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_7, l.output_size);

        l.kernel_size = l.width * l.height * l.nfilters * l.depth * sizeof(float);
        hipMalloc( (void**)&d_kernel_7, l.kernel_size);

        l.bias_size = l.nfilters * sizeof(float);
        hipMalloc((void**)&d_bias_7, l.bias_size);

        l.im_h=h;
        l.im_w=w;
        printf("\033[1;31mLOAD CONV2D_7: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case CONV2D_8:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_8, l.output_size);

        l.kernel_size = l.width * l.height * l.nfilters * l.depth * sizeof(float);
        hipMalloc( (void**)&d_kernel_8, l.kernel_size);

        l.bias_size = l.nfilters * sizeof(float);
        hipMalloc((void**)&d_bias_8, l.bias_size);

        l.im_h=h;
        l.im_w=w;
        printf("\033[1;31mLOAD CONV2D_8: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case MAXP2D_4:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_maxp_4, l.output_size);

        l.im_h=h;
        l.im_w=w;

        printf("\033[1;31mLOAD MAXP2D_4: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case CONV2D_9:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_9, l.output_size);

        l.kernel_size = l.width * l.height * l.nfilters * l.depth * sizeof(float);
        hipMalloc( (void**)&d_kernel_9, l.kernel_size);

        l.bias_size = l.nfilters * sizeof(float);
        hipMalloc((void**)&d_bias_9, l.bias_size);

        l.im_h=h;
        l.im_w=w;
        printf("\033[1;31mLOAD CONV2D_9: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case CONV2D_10:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_10, l.output_size);

        l.kernel_size = l.width * l.height * l.nfilters * l.depth * sizeof(float);
        hipMalloc( (void**)&d_kernel_10, l.kernel_size);

        l.bias_size = l.nfilters * sizeof(float);
        hipMalloc((void**)&d_bias_10, l.bias_size);

        l.im_h=h;
        l.im_w=w;
        printf("\033[1;31mLOAD CONV2D_10: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case UPSM2D_1:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_upsm_1, l.output_size);

        l.im_h=h;
        l.im_w=w;

        printf("\033[1;31mLOAD UPSM2D_1: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case CONV2D_11:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_11, l.output_size);

        l.kernel_size = l.width * l.height * l.nfilters * l.depth * sizeof(float);
        hipMalloc( (void**)&d_kernel_11, l.kernel_size);

        l.bias_size = l.nfilters * sizeof(float);
        hipMalloc((void**)&d_bias_11, l.bias_size);

        l.im_h=h;
        l.im_w=w;
        printf("\033[1;31mLOAD CONV2D_11: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case CONCAT_1:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_concat_1, l.output_size);

        l.im_h=h;
        l.im_w=w;

        printf("\033[1;31mLOAD CONCAT_1: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case CONV2D_12:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_12, l.output_size);

        l.kernel_size = l.width * l.height * l.nfilters * l.depth * sizeof(float);
        hipMalloc( (void**)&d_kernel_12, l.kernel_size);

        l.bias_size = l.nfilters * sizeof(float);
        hipMalloc((void**)&d_bias_12, l.bias_size);

        l.im_h=h;
        l.im_w=w;
        printf("\033[1;31mLOAD CONV2D_12: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case CONV2D_13:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_13, l.output_size);

        l.kernel_size = l.width * l.height * l.nfilters * l.depth * sizeof(float);
        hipMalloc( (void**)&d_kernel_13, l.kernel_size);

        l.bias_size = l.nfilters * sizeof(float);
        hipMalloc((void**)&d_bias_13, l.bias_size);

        l.im_h=h;
        l.im_w=w;
        printf("\033[1;31mLOAD CONV2D_13: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case UPSM2D_2:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_upsm_2, l.output_size);

        l.im_h=h;
        l.im_w=w;

        printf("\033[1;31mLOAD UPSM2D_2: image:%d,%d \n\033[0m",w,h);
        break;
    }

    case CONV2D_14:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_14, l.output_size);

        l.kernel_size = l.width * l.height * l.nfilters * l.depth * sizeof(float);
        hipMalloc( (void**)&d_kernel_14, l.kernel_size);

        l.bias_size = l.nfilters * sizeof(float);
        hipMalloc((void**)&d_bias_14, l.bias_size);

        l.im_h=h;
        l.im_w=w;
        printf("\033[1;31mLOAD CONV2D_14: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case CONCAT_2:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_concat_2, l.output_size);

        l.im_h=h;
        l.im_w=w;

        printf("\033[1;31mLOAD CONCAT_2: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case CONV2D_15:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_15, l.output_size);

        l.kernel_size = l.width * l.height * l.nfilters * l.depth * sizeof(float);
        hipMalloc( (void**)&d_kernel_15, l.kernel_size);

        l.bias_size = l.nfilters * sizeof(float);
        hipMalloc((void**)&d_bias_15, l.bias_size);

        l.im_h=h;
        l.im_w=w;
        printf("\033[1;31mLOAD CONV2D_15: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case CONV2D_16:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_16, l.output_size);

        l.kernel_size = l.width * l.height * l.nfilters * l.depth * sizeof(float);
        hipMalloc( (void**)&d_kernel_16, l.kernel_size);

        l.bias_size = l.nfilters * sizeof(float);
        hipMalloc((void**)&d_bias_16, l.bias_size);

        l.im_h=h;
        l.im_w=w;
        printf("\033[1;31mLOAD CONV2D_16: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case UPSM2D_3:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_upsm_3, l.output_size);

        l.im_h=h;
        l.im_w=w;

        printf("\033[1;31mLOAD UPSM2D_3: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case CONV2D_17:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_17, l.output_size);

        l.kernel_size = l.width * l.height * l.nfilters * l.depth * sizeof(float);
        hipMalloc( (void**)&d_kernel_17, l.kernel_size);

        l.bias_size = l.nfilters * sizeof(float);
        hipMalloc((void**)&d_bias_17, l.bias_size);

        l.im_h=h;
        l.im_w=w;
        printf("\033[1;31mLOAD CONV2D_17: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case CONCAT_3:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_concat_3, l.output_size);

        l.im_h=h;
        l.im_w=w;

        printf("\033[1;31mLOAD CONCAT_3: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case CONV2D_18:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_18, l.output_size);

        l.kernel_size = l.width * l.height * l.nfilters * l.depth * sizeof(float);
        hipMalloc( (void**)&d_kernel_18, l.kernel_size);

        l.bias_size = l.nfilters * sizeof(float);
        hipMalloc((void**)&d_bias_18, l.bias_size);

        l.im_h=h;
        l.im_w=w;
        printf("\033[1;31mLOAD CONV2D_18: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case CONV2D_19:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_19, l.output_size);

        l.kernel_size = l.width * l.height * l.nfilters * l.depth * sizeof(float);
        hipMalloc( (void**)&d_kernel_19, l.kernel_size);

        l.bias_size = l.nfilters * sizeof(float);
        hipMalloc((void**)&d_bias_19, l.bias_size);

        l.im_h=h;
        l.im_w=w;
        printf("\033[1;31mLOAD CONV2D_19: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case UPSM2D_4:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_upsm_4, l.output_size);

        l.im_h=h;
        l.im_w=w;

        printf("\033[1;31mLOAD UPSM2D_4: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case CONV2D_20:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_20, l.output_size);

        l.kernel_size = l.width * l.height * l.nfilters * l.depth * sizeof(float);
        hipMalloc( (void**)&d_kernel_20, l.kernel_size);

        l.bias_size = l.nfilters * sizeof(float);
        hipMalloc((void**)&d_bias_20, l.bias_size);

        l.im_h=h;
        l.im_w=w;
        printf("\033[1;31mLOAD CONV2D_20: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case CONCAT_4:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_concat_4, l.output_size);

        l.im_h=h;
        l.im_w=w;

        printf("\033[1;31mLOAD CONCAT_4: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case CONV2D_21:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_21, l.output_size);

        l.kernel_size = l.width * l.height * l.nfilters * l.depth * sizeof(float);
        hipMalloc( (void**)&d_kernel_21, l.kernel_size);

        l.bias_size = l.nfilters * sizeof(float);
        hipMalloc((void**)&d_bias_21, l.bias_size);

        l.im_h=h;
        l.im_w=w;
        printf("\033[1;31mLOAD CONV2D_20: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case CONV2D_22:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_22, l.output_size);

        l.kernel_size = l.width * l.height * l.nfilters * l.depth * sizeof(float);
        hipMalloc( (void**)&d_kernel_22, l.kernel_size);

        l.bias_size = l.nfilters * sizeof(float);
        hipMalloc((void**)&d_bias_22, l.bias_size);

        l.im_h=h;
        l.im_w=w;
        printf("\033[1;31mLOAD CONV2D_20: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case CONV2D_23:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_23, l.output_size);

        l.kernel_size = l.width * l.height * l.nfilters * l.depth * sizeof(float);
        hipMalloc( (void**)&d_kernel_23, l.kernel_size);

        l.bias_size = l.nfilters * sizeof(float);
        hipMalloc((void**)&d_bias_23, l.bias_size);

        l.im_h=h;
        l.im_w=w;
        printf("\033[1;31mLOAD CONV2D_20: image:%d,%d \n\033[0m",w,h);
        break;
    }
    case CONV2D_24:
    {
        l.output_size = w * h *l.nfilters * sizeof(float);
        hipMalloc(&d_output_24, l.output_size);

        l.kernel_size = l.width * l.height * l.nfilters * l.depth * sizeof(float);
        hipMalloc( (void**)&d_kernel_24, l.kernel_size);

        l.bias_size = l.nfilters * sizeof(float);
        hipMalloc((void**)&d_bias_24, l.bias_size);

        l.im_h=h;
        l.im_w=w;
        printf("\033[1;31mLOAD CONV2D_20: image:%d,%d \n\033[0m",w,h);
        break;
    }
    default:
        break;
    }


}

extern "C" void Remove_NN()
{
    //first layer
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_kernel);
    hipFree(d_bias);
    // second layer
    hipFree(d_output_2);
    hipFree(d_kernel_2);
    hipFree(d_bias_2);
    //third layer
    hipFree(d_output_maxp_1);
    // 4th layer
    hipFree(d_output_3);
    hipFree(d_kernel_3);
    hipFree(d_bias_3);
    // 5th layer
    hipFree(d_output_4);
    hipFree(d_kernel_4);
    hipFree(d_bias_4);
    // 6th layer
    hipFree(d_output_maxp_2);
    // 7th layer
    hipFree(d_output_5);
    hipFree(d_kernel_5);
    hipFree(d_bias_5);
    // 8th layer
    hipFree(d_output_6);
    hipFree(d_kernel_6);
    hipFree(d_bias_6);
    // 9th layer
    hipFree(d_output_maxp_3);
    // 10th layer
    hipFree(d_output_7);
    hipFree(d_kernel_7);
    hipFree(d_bias_7);
    // 11th layer
    hipFree(d_output_8);
    hipFree(d_kernel_8);
    hipFree(d_bias_8);
    // 12th layer
    hipFree(d_output_maxp_4);
    // 13th layer
    hipFree(d_output_9);
    hipFree(d_kernel_9);
    hipFree(d_bias_9);
    // 14th layer
    hipFree(d_output_10);
    hipFree(d_kernel_10);
    hipFree(d_bias_10);
    // 16th layer
    hipFree(d_output_upsm_1);
    // 17th layer
    hipFree(d_output_11);
    hipFree(d_kernel_11);
    hipFree(d_bias_11);
    // 18th layer
    hipFree(d_output_concat_1);
    // 20th layer
    hipFree(d_output_12);
    hipFree(d_kernel_12);
    hipFree(d_bias_12);
    // 21th layer
    hipFree(d_output_13);
    hipFree(d_kernel_13);
    hipFree(d_bias_13);
    // 22th layer
    hipFree(d_output_upsm_2);
    // 23th layer
    hipFree(d_output_14);
    hipFree(d_kernel_14);
    hipFree(d_bias_14);
    // 24th layer
    hipFree(d_output_concat_2);
    // 25th layer
    hipFree(d_output_15);
    hipFree(d_kernel_15);
    hipFree(d_bias_15);
    // 26th layer
    hipFree(d_output_16);
    hipFree(d_kernel_16);
    hipFree(d_bias_16);
    // 27th layer
    hipFree(d_output_upsm_3);
    // 28th layer
    hipFree(d_output_17);
    hipFree(d_kernel_17);
    hipFree(d_bias_17);
    // 29th layer
    hipFree(d_output_concat_3);
    // 30th layer
    hipFree(d_output_18);
    hipFree(d_kernel_18);
    hipFree(d_bias_18);
    // 31th layer
    hipFree(d_output_19);
    hipFree(d_kernel_19);
    hipFree(d_bias_19);
    // 32th layer
    hipFree(d_output_upsm_4);
    // 33th layer
    hipFree(d_output_20);
    hipFree(d_kernel_20);
    hipFree(d_bias_20);
    // 34th layer
    hipFree(d_output_concat_4);
    // 35th layer
    hipFree(d_output_21);
    hipFree(d_kernel_21);
    hipFree(d_bias_21);
    // 36th layer
    hipFree(d_output_22);
    hipFree(d_kernel_22);
    hipFree(d_bias_22);
    // 37th layer
    hipFree(d_output_23);
    hipFree(d_kernel_23);
    hipFree(d_bias_23);
    // 38th layer
    hipFree(d_output_24);
    hipFree(d_kernel_24);
    hipFree(d_bias_24);
    printf("\033[1;31mRemove weights from Memory\n\033[0m");
}
